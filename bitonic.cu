#include "hip/hip_runtime.h"
 /******************************************************************************
 * FILE: data_creation_cuda.cu
 * DESCRIPTION:
 *   This code will be used to create the 4 different types of data we want to
 *   sort on using CUDA threads.
 * AUTHOR: Roee Belkin, Ansley Thompson
 * LAST REVISED: 11/01/23
 ******************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* main_region = "main_region";
const char* data_init_region = "data_init_region";
const char* comm_region = "comm_region";
const char* comm_small_region = "comm_small_region";
const char* comm_large_region = "comm_large_region";
const char* correctness_check_region = "correctness_check_region";
const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

hipEvent_t main_time;
hipEvent_t bitonic_sort_step_start_time;
hipEvent_t bitonic_sort_step_end_time;
hipEvent_t host_to_device_start_time;
hipEvent_t host_to_device_end_time;
hipEvent_t device_to_host_start_time;
hipEvent_t device_to_host_end_time;

enum sort_type{
  SORTED,
  REVERSE_SORTED,
  PERTURBED,
  RANDOM
};


bool check_array(float* arr, int length){
  for (int i = 0; i < length -1; i++){
    if (arr[i] > arr[i+1]){
      return false;
    }
  }
  return true; 
}

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length, int sort_type)
{
  srand(time(NULL));
  int i;
  if (sort_type == RANDOM){
    for (i = 0; i < length; ++i) {
      arr[i] = random_float();
    } 
  }
  else if (sort_type == SORTED){
    for (i = 0; i < length; i++){
      arr[i] = i;
    }
  }
  else if (sort_type == PERTURBED){
    for(i = 0; i < length; i++){
      arr[i] = i;
      int temp = rand() % 100;
      if (temp == 1){
        arr[i] = rand() % length; 
      }
    }
  }
  else if (sort_type == REVERSE_SORTED){
    for (i = 0; i < length; i++){
      arr[i] = length - i - 1;
    }
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
int bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  
  //MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN(comm_large_region);
  hipEventRecord(host_to_device_start_time);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  hipEventRecord(host_to_device_end_time);
  CALI_MARK_END(comm_large_region);
  hipEventSynchronize(host_to_device_end_time);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  
  int j, k;
  int count = 0;
  /* Major step */
  CALI_MARK_BEGIN(bitonic_sort_step_region);
  hipEventRecord(bitonic_sort_step_start_time);
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      count++;
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipDeviceSynchronize();
  hipEventRecord(bitonic_sort_step_end_time);
  CALI_MARK_END(bitonic_sort_step_region);
  hipEventSynchronize(bitonic_sort_step_end_time);
  
  //MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN(comm_large_region);
  hipEventRecord(device_to_host_start_time);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipEventRecord(device_to_host_end_time);
  CALI_MARK_END(comm_large_region);
  hipFree(dev_values);
  hipEventSynchronize(device_to_host_end_time);

  printf("Count is: %d \n", count);
  return count;
}

int main(int argc, char *argv[])
{
  CALI_MARK_BEGIN(main_region);
  // hipEventCreate(&bitonic_sort_step_start_time);
  hipEventCreate(&bitonic_sort_step_end_time);
  hipEventCreate(&host_to_device_start_time);
  hipEventCreate(&host_to_device_end_time);
  hipEventCreate(&device_to_host_start_time);
  hipEventCreate(&device_to_host_end_time);

  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  // Create caliper ConfigManager object
  cali::ConfigManager mgr;
  mgr.start();

  clock_t start, stop;
  float c = 0;

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  CALI_MARK_BEGIN(data_init_region);
  array_fill(values, NUM_VALS, RANDOM);
  CALI_MARK_END(data_init_region);

  array_print(values, NUM_VALS); 

  start = clock();
  c = bitonic_sort(values); /* Inplace */
  stop = clock();

  print_elapsed(start, stop);

  array_print(values, NUM_VALS);
  if (!check_array(values, NUM_VALS)){
    printf("ERROR ARRAY IS NOT SORTED!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n\n\n\n");
  }

  // Store results in these variables.
  float effective_bandwidth_gb_s;
  float bitonic_sort_step_time;
  float cudaMemcpy_host_to_device_time;
  float cudaMemcpy_device_to_host_time;


  hipEventElapsedTime(&bitonic_sort_step_time, bitonic_sort_step_start_time, bitonic_sort_step_end_time);
  hipEventElapsedTime(&cudaMemcpy_host_to_device_time, host_to_device_start_time, host_to_device_end_time);
  hipEventElapsedTime(&cudaMemcpy_device_to_host_time, device_to_host_start_time, device_to_host_end_time);

  bitonic_sort_step_time /= 1000;
  cudaMemcpy_device_to_host_time /= 1000;
  cudaMemcpy_host_to_device_time /= 1000;
  float temp = (c*2*4*NUM_VALS) /(bitonic_sort_step_time);
  effective_bandwidth_gb_s = temp/1e9;

  CALI_MARK_END(main_region);

  printf("bitonic sort step time: %f \n", bitonic_sort_step_time);
  printf("host to device time: %f \n", cudaMemcpy_host_to_device_time);
  printf("device to host time: %f \n", cudaMemcpy_device_to_host_time);
  printf("effective bandwith time: %f \n", effective_bandwidth_gb_s);
  printf("Count is: %d \n", c);


  adiak::init(NULL);
  adiak::launchdate();    // launch date of the job
  adiak::libraries();     // Libraries used
  adiak::cmdline();       // Command line used to launch the job
  adiak::clustername();   // Name of the cluster
  adiak::value("Algorithm", "BitonicSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
  adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
  adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
  adiak::value("SizeOfDatatype", 4); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
  adiak::value("InputSize", 1024); // The number of elements in input dataset (1000)
  adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
  adiak::value("num_procs", "0"); // The number of processors (MPI ranks)
  adiak::value("num_threads", 2); // The number of CUDA or OpenMP threads
  adiak::value("num_blocks", 512); // The number of CUDA blocks 
  adiak::value("group_num", 14); // The number of your group (integer, e.g., 1, 10)
  adiak::value("implementation_source", "Lab"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").


  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
}